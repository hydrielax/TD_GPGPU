/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

 #include <iostream>
 #include "bitmap_image.hpp"

#define DIM 1000

struct hipComplex {
    float   r;
    float   i;
    hipComplex( float a, float b ) : r(a), i(b)  {}
    float magnitude2( void ) { return r * r + i * i; }
    hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

int julia( int x, int y ) { 
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(-0.8, 0.156);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

void kernel( unsigned char *ptr ){
    for (int y=0; y<DIM; y++) {
        for (int x=0; x<DIM; x++) {
            int offset = x + y * DIM;

            int juliaValue = julia( x, y );
            ptr[offset*3 + 0] = 255 * juliaValue;
            ptr[offset*3 + 1] = 0;
            ptr[offset*3 + 2] = 0;
        }
    }
 }

int main( void ) {
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    unsigned char *ptr = (unsigned char *)malloc(DIM*DIM*3);
    
    hipEventRecord(start, 0);
    kernel( ptr );
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to compute %3.1f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
	// Write BMP image
    bitmap_image img(DIM,DIM);
    img.clear();
    for (int y = DIM-1; y >= 0; y--)
	{
		for (int x = DIM-1; x >= 0; x--)
		{
            int offset = x + y * DIM;
            img.set_pixel(x, y, ptr[offset*3], ptr[offset*3+1], ptr[offset*3+2]);            
        }
    }
    img.save_image("test.bmp");
	return 0;
}

