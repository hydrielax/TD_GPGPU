
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512
#define NB_STREAMS 4
#define SEGMENT_SIZE (1024 * 128)

#define CREATE_CUDAEVENT     \
	hipEvent_t start, stop; \
	hipEventCreate(&start); \
	hipEventCreate(&stop);

#define START_CUDAEVENT hipEventRecord(start, 0);
#define STOP_AND_PRINT_CUDAEVENT(txt)                       \
	hipEventRecord(stop, 0);                               \
	hipEventSynchronize(stop);                             \
	{                                                       \
		float elapsedTime;                                  \
		hipEventElapsedTime(&elapsedTime, start, stop);    \
		printf("Time to %s %3.1f ms\n", #txt, elapsedTime); \
	}

__global__ void vector_add(int *a, int *b, int *c)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	c[index] = a[index] + b[index];
}

void stream_addition(int *a, int *b, int *c)
{
	CREATE_CUDAEVENT
	int *d_a[NB_STREAMS];
	int *d_b[NB_STREAMS];
	int *d_c[NB_STREAMS];
	hipStream_t streams[NB_STREAMS];
	// int size = N * sizeof(int) / NB_STREAMS;

	for (int i = 0 ; i < NB_STREAMS ; i++) {
		hipStreamCreate(&(streams[i]));
		hipMalloc((void **)&(d_a[i]), SEGMENT_SIZE * sizeof(int));
		hipMalloc((void **)&(d_b[i]), SEGMENT_SIZE * sizeof(int));
		hipMalloc((void **)&(d_c[i]), SEGMENT_SIZE * sizeof(int));
	}

	START_CUDAEVENT
	
	for (int i = 0 ; i < N ; i += SEGMENT_SIZE * NB_STREAMS) {

		for (int s = 0 ; s < NB_STREAMS ; s++) {
			hipMemcpyAsync(
				d_a[s],
				a + i + SEGMENT_SIZE * s,
				SEGMENT_SIZE * sizeof(int),
				hipMemcpyHostToDevice,
				streams[s]
			);
			hipMemcpyAsync(
				d_b[s],
				b + i + SEGMENT_SIZE * s,
				SEGMENT_SIZE * sizeof(int),
				hipMemcpyHostToDevice,
				streams[s]
			);
		}

		for (int s = 0 ; s < NB_STREAMS ; s++) {
			vector_add<<<SEGMENT_SIZE / THREADS_PER_BLOCK, THREADS_PER_BLOCK, 0, streams[s]>>>(d_a[s], d_b[s], d_c[s]);
		}

		for (int s = 0 ; s < NB_STREAMS ; s++) {
			hipMemcpyAsync(
				c + i + SEGMENT_SIZE * s,
				d_c[s],
				SEGMENT_SIZE * sizeof(int),
				hipMemcpyDeviceToHost,
				streams[s]
			);
		}
		
	}
	
	for (int s = 0 ; s < NB_STREAMS ; s++) {
		hipStreamDestroy(streams[s]);
	}
	STOP_AND_PRINT_CUDAEVENT(computation)
	
	/* clean up */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

void addition(int *a, int *b, int *c)
{
	CREATE_CUDAEVENT
	int size = N * sizeof(int);
	int *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	START_CUDAEVENT
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	STOP_AND_PRINT_CUDAEVENT(memcpy h2d)

	START_CUDAEVENT
	vector_add<<<(N + (THREADS_PER_BLOCK - 1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
	STOP_AND_PRINT_CUDAEVENT(computation)

	START_CUDAEVENT
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	STOP_AND_PRINT_CUDAEVENT(memcpy d2h)

	/* clean up */
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}

int main()
{
	int *a, *b, *c;
	int size = N * sizeof(int);

	/* Pinned memory */
	hipHostAlloc((void **)&a, size, hipHostMallocDefault);
	hipHostAlloc((void **)&b, size, hipHostMallocDefault);
	hipHostAlloc((void **)&c, size, hipHostMallocDefault);

	for (int i = 0; i < N; i++)
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	printf("Addition with default stream\n");
	addition(a, b, c);

	printf("c[0] = %d\n", c[0]);
	printf("c[%d] = %d\n", N - 1, c[N - 1]);

	// Reinitialisation
	for (int i = 0; i < N; i++)
	{
		c[i] = 0;
	}

	/*< Add a call to your function with streams >*/
	printf("Addition with streams\n");
	stream_addition(a, b, c);

	printf("c[0] = %d\n", c[0]);
	printf("c[%d] = %d\n", N - 1, c[N - 1]);

	/* clean up */
	hipHostFree(a);
	hipHostFree(b);
	hipHostFree(c);

	return 0;
}
