#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include "text.h"
#include <iostream>

#define NB_ASCII_CHAR 128

const int threadsPerBlock = 256;

__global__ void kernel(char *buffer, int size, u_int *histo) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    int stride = blockDim.x * gridDim.x;

    while (i < size) {
        atomicAdd(&(histo[buffer[i]]), 1);
        i += stride;
    }
}

int main(void)
{
    int len = strlen(h_str);
    printf("len:%d\n", len);
    int size = len * sizeof(char);

    // GPU COMPUTATION

    // create variables
    char *d_str;
    u_int *histo;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // allocate & copy
    hipMalloc( (void**) &d_str, size );
    hipMemcpy(d_str, h_str, size, hipMemcpyHostToDevice);

    // share memory
    hipMallocManaged(&histo, NB_ASCII_CHAR * sizeof(u_int));

    // nb blocks
    const int nb_blocks = (len + threadsPerBlock - 1) / threadsPerBlock;

    // call kernel
    kernel<<<nb_blocks, threadsPerBlock>>>(d_str, len, histo);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Total time for %d elements was %f ms\n", len, elapsedTime);

    // PRINT
    for (int bean = 0; bean < NB_ASCII_CHAR; bean++)
    {
        std::cout << (char)bean << " : " << histo[bean] << std::endl;
    }

    hipFree(histo);
    hipFree(d_str);

    return 0;
}
