/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

 
#include <hip/hip_runtime.h>
#include <stdio.h>

 
 #define N 4000000
 #define RADIUS 5
 #define THREADS_PER_BLOCK 512
 
 __global__
 void stencil_1d(int n, double *in, double *out)
 {
  /* calculate global index in the array */
  /* insert code to calculate global index in the array using block
  and thread built-in variables */
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
     
  /* return if my global index is larger than the array size */
  if( gindex < n ){
 
   /* code to handle the boundary conditions */
   if( gindex < RADIUS || gindex >= (n - RADIUS) ) 
   {
     out[gindex] = (double) gindex * ( (double)RADIUS*2 + 1) ;
   }
   else{ 
     double result = 0.0;
     
     for( int i = gindex-(RADIUS); i <= gindex+(RADIUS); i++ ) 
     {
       /* add the required elements from the array "in" to the temporary 
       variable "result */
       result += in[i];
     } 
     out[gindex] = result;
    }
  }
 
 }
 
 int main()
 {
   double *in, *out;
   double *d_in, *d_out;
   int size = N * sizeof( double );
 
 /* allocate space for device copies of in, out */
 
   hipMalloc( (void **) &d_in, size );
   hipMalloc( (void **) &d_out, size );
 
 /* allocate space for host copies of in, out and setup input values */
 
   in = (double *)malloc( size );
   out = (double *)malloc( size );
 
   for( int i = 0; i < N; i++ )
   {
     in[i] = (double) i;
     out[i] = 0;
   }
 
 /* copy inputs to device */
 
   hipMemcpy( d_in, in, size, hipMemcpyHostToDevice );
   hipMemset( d_out, 0, size );
 
 /* calculate block and grid sizes */
 
   dim3 threads( THREADS_PER_BLOCK, 1, 1);
 /* insert code for proper number of blocks in X dimension */
   dim3 blocks( N / THREADS_PER_BLOCK+1, 1, 1);
 
 /* start the timers */
 
   hipEvent_t start, stop;
   hipEventCreate( &start );
   hipEventCreate( &stop );
   hipEventRecord( start, 0 );
 
 /* launch the kernel on the GPU */
 
   stencil_1d<<< blocks, threads >>>( N, d_in, d_out );
   
   hipDeviceSynchronize();
 
 /* stop the timers */
 
   hipEventRecord( stop, 0 );
   hipEventSynchronize( stop );
   float elapsedTime;
   hipEventElapsedTime( &elapsedTime, start, stop );
 
   printf("Total time for %d elements was %f ms\n", N, elapsedTime );
 
 /* copy result back to host */
 
   hipMemcpy( out, d_out, size, hipMemcpyDeviceToHost );
 
   for( int i = 0; i < N; i++ )
   {
     if( in[i]*( (double)RADIUS*2+1 ) != out[i] ) 
     {
       printf("error in element %d in = %f out %f\n",i,in[i],out[i] );
       printf("FAIL\n");
       goto end;
     } /* end if */
   } /* end for */
 
   printf("PASS\n");
   end:
 
 /* clean up */
 
   free(in);
   free(out);
   hipFree( d_in );
   hipFree( d_out );
 
   hipDeviceReset();
     
   return 0;
 } /* end main */