
#include <hip/hip_runtime.h>
#include <stdio.h>

/* experiment with N */
/* how large can it be? */
#define N (2047 * 2047)
#define THREADS_PER_BLOCK 512

__global__ void vector_add(int *a, int *b, int *c)
{
	/* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = threadIdx.x + blockDim.x * blockIdx.x;
	if (index < N)
	{
		c[index] = a[index] + b[index];
	}
}

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	/* record start time */
	hipEventRecord(start, 0);

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int size = N * sizeof(int);

	/* allocate space for device copies of a, b, c */
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);

	for (int i = 0; i < N; i++)
	{
		a[i] = b[i] = i;
		// c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	/* take the number of blocks */
	int nb_blocks = ceil((float)N / THREADS_PER_BLOCK);

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */
	vector_add<<<nb_blocks, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	printf("c[%d] = %d\n", 0, c[0]);
	printf("c[%d] = %d\n", 5, c[5]);
	printf("c[%d] = %d\n", N - 1, c[N - 1]);

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/* record finish time */
	hipEventRecord(stop, 0);
	/* wait GPU event */
	hipEventSynchronize(stop);

	/* compute and print ellapsed time between start and stop */
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to execute %3.1f ms\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
} /* end main */
